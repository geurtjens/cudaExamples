#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void helloWorldFromThread() {
	printf("Hello World, blockIdx.x: %d, threadIdx.x: %d\n", blockIdx.x,threadIdx.x);
}

int main()
{
	helloWorldFromThread << <3, 4 >> > ();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}